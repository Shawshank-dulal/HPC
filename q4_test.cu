#include "hip/hip_runtime.h"
    #include <stdio.h>
    #include <stdio.h>
    #include <stdlib.h>
    #include "lodepng.cpp"
    #include <hip/hip_runtime_api.h>


    __global__ void cudaBlur(unsigned char *imageInput, unsigned char *imageOutput, unsigned int height, unsigned int width)
    {
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        int pixel = idx * 4;
        float red = 0;
        float green = 0;
        float blue = 0;
        float alpha = 0;
         int count = 1;

        int top=(idx/width)-1;
        int left=(idx%width)-1;
        int right=(idx%width)+1;
        int bottom=(idx/width)+1;

        red += imageInput[pixel];
        green += imageInput[1 + pixel];
        blue += imageInput[2 + pixel];
        alpha += imageInput[3 + pixel];
        count++;

        if(bottom && bottom<=height && left>0 ){
            int currentPos = width * (idx / width + 1) + idx % width - 1;
            int pixel = currentPos * 4;
            red += imageInput[pixel];
            green += imageInput[1 + pixel];
            blue += imageInput[2 + pixel];
            alpha += imageInput[3 + pixel];
            count++;
        }
        if(right && right<=width){
            int currentPos = width * (idx / width) + idx % width + 1;
            int pixel = currentPos * 4;
            red += imageInput[pixel];
            green += imageInput[1 + pixel];
            blue += imageInput[2 + pixel];
            alpha += imageInput[3 + pixel];
            count++;
        }
        if(bottom && bottom<=height && right<=width){
            int currentPos = width * (idx / width) + idx % width + 1;
            int pixel = currentPos * 4;
            red += imageInput[pixel];
            green += imageInput[1 + pixel];
            blue += imageInput[2 + pixel];
            alpha += imageInput[3 + pixel];
            count++;
        }
        if(bottom && bottom<=height){
            int currentPos = width * (idx / width + 1) + idx % width;
            int pixel = currentPos * 4;
            red += imageInput[pixel];
            green += imageInput[1 + pixel];
            blue += imageInput[2 + pixel];
            alpha += imageInput[3 + pixel];
            count++;
        }
        if(left && left>0){
            int currentPos = width * (idx / width) + idx % width - 1;
            int pixel = currentPos * 4;
            red += imageInput[pixel];
            green += imageInput[1 + pixel];
            blue += imageInput[2 + pixel];
            alpha += imageInput[3 + pixel];
            count++;
        }
        if(top && top>0){
            int currentPos = width * (idx / width - 1) + idx % width;
            int pixel = currentPos * 4;
            red += imageInput[pixel];
            green += imageInput[1 + pixel];
            blue += imageInput[2 + pixel];
            alpha += imageInput[3 + pixel];
            count++;
        }

        int currentIdx = idx * 4;
        imageOutput[currentIdx] = red / count;
        imageOutput[currentIdx + 1] = green / count;
        imageOutput[currentIdx + 2] = blue / count;
        imageOutput[currentIdx + 3] = imageInput[3 + currentIdx];
    }

    int main()
    {
        unsigned int error;
        unsigned int encError;
        unsigned char* image;
        unsigned int width;
        unsigned int height;
        unsigned int *d_height, *d_width;
        const char *filename = "hck.png";
        const char *newFileName = "generated.png";

        error = lodepng_decode32_file(&image, &width, &height, filename);
        if (error)
        {
            printf("error %u: %s\n", error, lodepng_error_text(error));
        }

        const int ARRAY_SIZE = width * height * 4;
        const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

        unsigned char host_imageInput[ARRAY_SIZE * 4];
        unsigned char host_imageOutput[ARRAY_SIZE * 4];

        for (int i = 0; i < ARRAY_SIZE; i++)
        {
            host_imageInput[i] = image[i];
        }

        unsigned char *d_in;
        unsigned char *d_out;

        hipMalloc((void **)&d_in, ARRAY_BYTES);
        hipMalloc((void **)&d_out, ARRAY_BYTES);

        hipMalloc(&d_height, sizeof(height));
        hipMalloc(&d_width, sizeof(width));

        hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

        hipMemcpy(d_height, &height, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(d_width, &width, sizeof(unsigned int), hipMemcpyHostToDevice);

        cudaBlur<<<height, width>>>(d_in, d_out, height, width);

        hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

        encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
        if (encError)
        {
            printf("error %u: %s\n", error, lodepng_error_text(encError));
        }

        hipFree(d_in);
        hipFree(d_out);
        printf("completed");
        return 0;
    }