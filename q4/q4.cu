#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.cpp"
#include<hip/hip_runtime_api.h>
// #include<hip/hip_runtime_api.h>
/*This Code uses gaussian blurring technique to blurr the image. In order to compile the code, 
run "gcc task4.c -o <output filename> -pthread lodepng.c"
and to run the code, run "./<output filename>"*/
unsigned int error, encError;
unsigned char *imageInput, *imageOutput;
unsigned int width;
unsigned int height;

__global__  void cudaBlur(unsigned char *imageInput)
{
    float t_r = 0, t_g = 0, t_b = 0, t_a = 0, counter = 1;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx*4;
    t_r += imageInput[pixel];
    t_g += imageInput[1 + pixel];
    t_b += imageInput[2 + pixel];
    t_a += imageInput[3 + pixel];
    counter++;
    if (i + 1 && j - 1)
    {
        int pos = pixel;
        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (j + 1)
    {
        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (i + 1 && j + 1)
    {

        t_r += imageInput[pixel];index
        counter++;
    }
    if (i + 1)
    {

        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (j - 1)
    {

        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (i - 1)
    {

        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    int current_pixel = idx * 4;
    imageOutput[current_pixel] = t_r / counter;
    imageOutput[current_pixel + 1] = t_g / counter;
    imageOutput[current_pixel + 2] = t_b / counter;
    imageOutput[current_pixel + 3] = imageInput[3 + current_pixel];
}


void *blurring(void *p)
{
    unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "hck.png";
	const char* newFileName = "generated.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	cudaBlur<<<height, width>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}

	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}