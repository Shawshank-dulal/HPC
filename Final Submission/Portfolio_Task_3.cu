
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime_api.h>

#define bool int
#define true 1
#define false 0

/* This program runs on CUDA platform to decrypt the encrypted password that we get from the CUDACrypt program
after the CUDA processor finally finds the password for the provided encryption, it returns the password to the main program and
it is printed on the screen from there.
In order to run the program first compile it by command: nvcc Portfolio_Task_3.cu -o <output_filename>
then run it by the command: ./<output_filename <encrypted_password>
you can find some encrypted passwords below

 sample passwords:
E X 5 3 = GCFBUW7371
B Y 2 1 = DBCCVX4053
W X 4 2 = YUXBUW6262
N X 0 0 = PLOBUW2244
X Y 4 3 = ZVYCVX6271
X Y 9 4 = ZVYCVX2780
 */


//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables

__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);
 
	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 90){
				newPassword[i] = (newPassword[i] - 90) + 65;
			}else if(newPassword[i] < 65){
				newPassword[i] = (65 - newPassword[i]) + 65;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}
__device__ bool cudaCmp(char* s1,char* s2){
    for(int i=0;i<10;i++){
        if(s1[i]!=s2[i])
            return false;
    }
    return true;

}

__global__ void crack(char * alphabet, char * numbers, char* encrypted,char * d_crackedPass){
char genRawPass[4];
genRawPass[0] = alphabet[blockIdx.x];
genRawPass[1] = alphabet[blockIdx.y];
genRawPass[2] = numbers[threadIdx.x];
genRawPass[3] = numbers[threadIdx.y];

char* generatedEncryption=CudaCrypt(genRawPass);
//printf("Pass: %s enc:%s\n",genRawPass, generatedEncryption);

if(cudaCmp(encrypted,generatedEncryption)){
	for(int i=0;i<4;i++){
		d_crackedPass[i]=genRawPass[i];
	}
}
// printf("%c %c %c %c = %s\n", genRawPass[0],genRawPass[1],genRawPass[2],genRawPass[3], CudaCrypt(genRawPass));

}

int main(int argc, char ** argv){
char encryptedPassword[10];
strcpy(encryptedPassword,argv[1]);
char cpuAlphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};
char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};
char *d_encryptedPass;
char * gpuAlphabet;
char *d_crackedPass;
char *h_crackedPass=(char *)malloc(sizeof(char)*4); 
hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
hipMalloc((void**) &d_crackedPass, sizeof(char) * 4);
hipMalloc( (void**) &d_encryptedPass, sizeof(encryptedPassword)); 
hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);
hipMemcpy(d_encryptedPass, encryptedPassword, sizeof(encryptedPassword), hipMemcpyHostToDevice);

char * gpuNumbers;
hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26); 
hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers,d_encryptedPass,d_crackedPass);
hipDeviceSynchronize();
hipMemcpy(h_crackedPass, d_crackedPass, sizeof(char) * 4, hipMemcpyDeviceToHost);
printf("Password Found: %s\n",h_crackedPass);
return 0;
}
