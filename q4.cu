#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.cpp"
#include<hip/hip_runtime_api.h>

unsigned int error, encError;
unsigned char *imageInput, *imageOutput;
unsigned int width;
unsigned int height;

__global__  void cudaBlur(unsigned char *imageInput,unsigned char *imageOutput,unsigned int height,unsigned int width)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	  int pixel = idx*4;
    int current_w=idx%width;
    int current_h=(idx/width);
    int i = current_h, j = current_w;
 printf("index: %d current W= %d current h= %d\n",idx, current_w,current_h);
    float t_r = 0, t_g = 0, t_b = 0, t_a = 0, counter = 1;
    
    t_r += imageInput[pixel];
    t_g += imageInput[1 + pixel];
    t_b += imageInput[2 + pixel];
    t_a += imageInput[3 + pixel];
    counter++;
    if (i + 1<=height && j - 1>0 )
    {
        int pos = width * (current_h + 1) + current_w - 1;
        int pixel = pos * 4;
        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (j + 1<=width)
    {
        int pos = width * (current_h) + current_w + 1;
        int pixel = pos * 4;
        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (i + 1<=height && j + 1<=width)
    {
        int pos = width * (current_h + 1) + current_w + 1;
        int pixel = pos * 4;
        t_r += imageInput[pixel];
        counter++;
    }
    if (i + 1<=height)
    {
      int pos = width * (current_h + 1) + current_w;
        int pixel = pos * 4;
        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (j - 1>0)
    {
      int pos = width * (current_h) + current_w - 1;
        int pixel = pos * 4;
        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
    if (i - 1>0 )
    {
        int pos = width * (current_h - 1) + current_w;
        int pixel = pos * 4;
        t_r += imageInput[pixel];
        t_g += imageInput[1 + pixel];
        t_b += imageInput[2 + pixel];
        t_a += imageInput[3 + pixel];
        counter++;
    }
  
    int current_pixel = idx * 4;
    imageOutput[current_pixel] = t_r / counter;
    imageOutput[current_pixel + 1] = t_g / counter;
    imageOutput[current_pixel + 2] = t_b / counter;
    imageOutput[current_pixel + 3] = imageInput[3 + current_pixel];
}


int main()
{
    unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
 unsigned int *d_height, *d_width;
	const char* filename = "hck.png";
	const char* newFileName = "generated.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	unsigned char * d_in;
	unsigned char * d_out;

	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);
  

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	cudaBlur<<<height, width>>>(d_in, d_out,height,width);

	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}

	
	hipFree(d_in);
	hipFree(d_out);
 printf("completed");
 return 0;

}